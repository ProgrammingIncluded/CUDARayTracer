#include "hip/hip_runtime.h"
#include "OpenGLTest.cuh"

__global__ void createGPUTexture(uchar4* d_texture)
{
	uint pixelID = blockIdx.x*blockDim.x + threadIdx.x;
	
	uchar4 value;
	value.x = 150;
	value.y = 0;
	value.z = 0;
	value.w = 0;
	d_texture[pixelID].x = 125;
	d_texture[pixelID].y = 0;
	d_texture[pixelID].z =  0;
	d_texture[pixelID].w = 0;
}
__global__ void wow(uchar4* pos, unsigned int width, unsigned int height,
	float time)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int x = index%width;
	unsigned int y = index / width;

	if (index < width*height) {
		unsigned char r = (x + (int)time) & 0xff;
		unsigned char g = (y + (int)time) & 0xff;
		unsigned char b = ((x + y) + (int)time) & 0xff;

		// Each thread writes one pixel location in the texture (textel)
		pos[index].w = 0;
		pos[index].x = r;
		pos[index].y = g;
		pos[index].z = b;
	}
}
void drawFrame()
{
	glColor3f(1.0,1.0,1.0);
	glBindTexture(GL_TEXTURE_2D, gltexture);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, windowSize.x, windowSize.y, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	glBegin(GL_QUADS);
	glTexCoord2f(0.0f, 0.0f);
	glVertex2f(0.0f, 0.0f);
	glTexCoord2f(1.0f, 0.0f);
	glVertex2f(float(windowSize.x), 0.0f);
	glTexCoord2f(1.0f, 1.0f);
	glVertex2f(float(windowSize.x), float(windowSize.y));
	glTexCoord2f(0.0f, 1.0f);
	glVertex2f(0.0f, float(windowSize.y));
	glEnd();

	// Release
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
	glBindTexture(GL_TEXTURE_2D, 0);
}

void createFrame(float time)
{
	hipGraphicsMapResources(1, &cudaPBO, 0);
	size_t numBytes;
	hipGraphicsResourceGetMappedPointer((void**)&d_textureBufferData, &numBytes, cudaPBO);
	
	int totalThreads = windowSize.x * windowSize.y;
	int nBlocks = totalThreads/ 256;

	// Run code here.
	//createGPUTexture << <nBlocks,  256>> >(d_textureBufferData);
	wow << <nBlocks, 256 >> >(d_textureBufferData, windowSize.x, windowSize.y, time);
	// Unmap mapping to PBO so that OpenGL can access.
	hipGraphicsUnmapResources(1, &cudaPBO, 0);
}

void setupOpenGL()
{
	image  = new uchar4[256*256];

	glViewport(0, 0, windowSize.x, windowSize.y);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, windowSize.x, windowSize.y, 0.0, -1.0, 1.0);

	// Unbind any textures from previous.
	glBindTexture(GL_TEXTURE_2D, 0);

	// Create new textures.
	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &gltexture);
	glBindTexture(GL_TEXTURE_2D, gltexture);

	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	// Create image with same resolution as window.
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, windowSize.x , windowSize.y, 0, GL_RGBA, GL_UNSIGNED_BYTE, image);


	// Create pixel buffer boject.
	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, windowSize.x * windowSize.y * sizeof(uchar4), image, GL_STREAM_COPY);

	hipGraphicsGLRegisterBuffer(&cudaPBO, pbo, cudaGraphicsMapFlagsWriteDiscard);
	
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
	glBindTexture(GL_TEXTURE_2D, 0);
}