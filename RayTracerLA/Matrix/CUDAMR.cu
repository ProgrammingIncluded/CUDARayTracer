#include "hip/hip_runtime.h"
#include "CUDAMR.cuh"

/* Cuda Matrix Related Operations */



/**
* Subtracts two CUDAMR objects together given their values.
*/
__global__ void kernelSubCUDAMR(float* mA, float* mB, uint vectorSize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// Safety if statement.
	if (index < vectorSize)
	{
		mA[index] -= mB[index];
	}
}

CUDA_MFUNCTION void subCUDAMR(float* mA, float* mB, uint vectorSize)
{
	for (uint index = 0; index < vectorSize; ++index)
		mA[index] -= mB[index];
}

/**
* Multiplies one matrix by another matrix. Assumes square.
*/
__global__ void kernelMultCUDAMR(float* A, float* B, float* C, uint aRow, uint bRow, uint cRow) {
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < aRow; ++e)
		Cvalue += A[row * aRow + e] *
		B[e * bRow + col];
	C[row * cRow + col] = Cvalue;
}

CUDA_MFUNCTION void multCUDAMR(float* A, float* B, float* C, uint aRow, uint bRow, uint cRow)
{
	float Cvalue = 0;
	for (uint row = 0; row < aRow; ++row){
		for (uint col = 0; row < bRow; ++row){
			for (int e = 0; e < aRow; ++e)
				Cvalue += A[row * aRow + e] *
				B[e * bRow + col];
			C[row * cRow + col] = Cvalue;
		}
	}
}

/**
* Adds two CUDAMR objects together given their values.
*/
__global__ void kernelAddCUDAMR(float* mA, float* mB, uint vectorSize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// Safety if statement.
	if (index < vectorSize)
	{
		mA[index] += mB[index];
	}
}

CUDA_MFUNCTION void addCUDAMR(float* mA, float* mB, uint vectorSize)
{
	for (uint index = 0; index < vectorSize; ++index)
		mA[index] += mB[index];
}

namespace mat
{
	CUDAMR::CUDAMR(uint x, uint y)
	{
		if (x < 0)
			x = -x;
		if (y < 0)
			y = -y;

		value = nullptr;
		d_value = nullptr;

		size = sf::Vector2u(x, y);
		byteSize = size.x * size.y *sizeof(float);
		value = (float*)malloc(byteSize);
	}

	CUDAMR::~CUDAMR()
	{
		if (value != NULL)
			free(value);
		value = NULL;
		deallocateGPUMemory();
	}

	bool CUDAMR::setValue(float values[], uint size)
	{
		if (size != this->size.x*this->size.y)
			return false;

		std::memcpy(this->value, values, sizeof(float)*size);
		return true;
	}

	void CUDAMR::copyGPUValue()
	{
		if (this->d_value == nullptr)
			return;
		//hipDeviceSynchronize();
		// You have to copy the result or use friends.
		hipMemcpy(this->value, this->d_value, byteSize, hipMemcpyDeviceToHost);
	}

	void CUDAMR::copyValueToGPU()
	{
		if (this->d_value == nullptr)
		return;

		hipMemcpy(this->d_value, this->value, byteSize, hipMemcpyHostToDevice);
	}

	bool CUDAMR::allocateGPUMemory()
	{
		if (d_value != nullptr)
			return false;

		// Allocate!
		hipMalloc(&d_value, byteSize);
		return true;
	}


	bool CUDAMR::deallocateGPUMemory()
	{
		if (d_value == nullptr)
			return false;
		hipFree(d_value);
		return true;
	}

	void CUDAMR::add(CUDAMR* value)
	{
		if (value->size.x != this->size.x || value->size.y != this->size.y)
			return;

		// Check if GPU memory is allocated.
		if (this->d_value == nullptr || value->d_value == nullptr)
			return; // Add code to allocate or just silently return?

		// Kelvyne++
		kernelAddCUDAMR << <1, size.x*size.y >> > (d_value, value->d_value, size.x*size.y);
	}

	void CUDAMR::sub(CUDAMR* value)
	{
		if (value->size.x != this->size.x || value->size.y != this->size.y)
			return;

		// Check if GPU memory is allocated.
		if (this->d_value == nullptr || value->d_value == nullptr)
			return; // Add code to allocate or just silently return?

		// Kelvyne++
		kernelSubCUDAMR << <1, size.x*size.y >> > (d_value, value->d_value, size.x*size.y);
	}

	void CUDAMR::mult(CUDAMR* value, CUDAMR* result)
	{
		if (value->size.x != this->size.y || result->size.x != this->size.x || result->size.y != value->size.y)
			return;
		// Check if GPU memory is allocated.
		if (this->d_value == nullptr || value->d_value == nullptr || result->d_value == nullptr)
			return; // Add code to allocate or just silently return?

		dim3 grid(1, 1);
		dim3 thread(size.x, size.y);

		kernelMultCUDAMR << <grid, thread >> > (d_value, value->d_value, result->d_value, size.y, 
			value->size.y, result->size.y);
	}

	std::ostream& operator<<(std::ostream& os, const CUDAMR& obj)
	{

		for (int x = 0; x < obj.size.x; ++x)
		{
			for (int i = 0; i < obj.size.y; ++i)
			{
				os << obj.value[x * obj.size.y + i];
				os << " ";
			}
			os << std::endl;
		}
		return os;
	}

}